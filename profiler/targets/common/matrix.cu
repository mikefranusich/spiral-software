/*
 *  Copyright (c) 2018-2021, Carnegie Mellon University
 *  See LICENSE for details
 */
/***************************************************************************
 * SPL Matrix                                                              *
 *                                                                         *
 * Computes matrix that corresponds to SPL generated routine               *
 ***************************************************************************/

#include <limits.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>

#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

#include <hip/hip_runtime_api.h>

#ifndef ROWS
#error ROWS must be defined
#endif
#ifndef COLUMNS
#error COLUMNS must be defined
#endif

#ifndef NZERO
#define NZERO (1.0/(double)-INFINITY)
#endif

hipfftDoubleReal  *Input, *Output;
hipfftDoubleReal  *dev_in, *dev_out;

void initialize(int argc, char **argv) {
	Input =  (hipfftDoubleReal*) calloc(sizeof(hipfftDoubleReal), COLUMNS );
	Output = (hipfftDoubleReal*) calloc(sizeof(hipfftDoubleReal), ROWS );

	hipMalloc     ( &dev_in,  sizeof(hipfftDoubleReal) * COLUMNS );
	hipMalloc     ( &dev_out, sizeof(hipfftDoubleReal) * ROWS );

	INITFUNC();
}

void finalize() {
	free (Output);
	free (Input);
	hipFree     (dev_out);
	hipFree     (dev_in);
}

void set_value_in_vector(hipfftDoubleReal *arr, int elem)
{
	// Zero array and put '1' in the location indicated by element
	int idx;
	for (idx = 0; idx < COLUMNS; idx++)
		arr[idx] = (idx == elem) ? 1.0 : 0.0;

	return;
}

void compute_matrix()
{
	int x, y, indx;
	double nzero = NZERO;
	printf("[ ");
	for (x = 0; x < COLUMNS; x++) {
		set_value_in_vector(Input, x);

		hipMemcpy ( dev_in, Input, sizeof(hipfftDoubleReal) * COLUMNS, hipMemcpyHostToDevice);
		
		for (indx = 0; indx < ROWS; indx++) {
			Output[indx] = (double)-INFINITY;
			hipMemcpy(&dev_out[indx], &nzero, sizeof(hipfftDoubleReal), hipMemcpyHostToDevice);
			checkCudaErrors(hipGetLastError());
		}
		
		FUNC(dev_out, dev_in);
		hipMemcpy ( Output, dev_out, sizeof(hipfftDoubleReal) * ROWS, hipMemcpyDeviceToHost);
		
		if (x != 0) {
			printf(",\n  [ ");
		}
		else {
			printf("[ ");
		}
		for (y = 0; y < ROWS; y++) {
			if (y != 0) {
				if ((y % 10) == 0) {
					printf("\n");
				}
				printf(", ");
			}
			printf("FloatString(\"%.18g\")", Output[y]);
		}
		printf(" ]");
	}
	printf("\n];\n");
}


int main(int argc, char** argv) {
	initialize(argc, argv);
	compute_matrix();
	finalize();
	return EXIT_SUCCESS;
}
